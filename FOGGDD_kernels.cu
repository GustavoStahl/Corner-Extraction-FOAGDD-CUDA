#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <npp.h>

#include "hip/hip_runtime_api.h"
#include "helper_npp.h"

#define DIRECTIONS_MAX 8
#define FILTER_MAX 7
#define BLOCK_SIZE 25

/* 
   ##.##
   #...#
   ..... = 12 corners
   #...#
   ##.## 
*/
#define CORNER_NUM 12
#define MASK_MAX FILTER_MAX*FILTER_MAX - CORNER_NUM

__device__ float determinant(float triang_mat[][DIRECTIONS_MAX], const size_t rank)
{
    float ratio, det=1.f;

    /* float triang_mat[DIRECTIONS_MAX][DIRECTIONS_MAX];
    for(size_t i=0; i<rank; i++)
        for(size_t j=0; j<rank; j++)
            triang_mat[i][j] = matrix[i][j]; */

    for(size_t i=0; i<rank; i++)
    {
        // if(triang_mat[i][i] == 0.0) {printf("Mathematical Error!");}
        
        for(size_t j=i+1; j<rank; j++)
        {
            ratio = triang_mat[j][i]/triang_mat[i][i];

            for(size_t k=0; k<rank; k++)
            {
                triang_mat[j][k] = triang_mat[j][k] - ratio*triang_mat[i][k];
            }
        }
    }

    for(size_t i=0; i<rank; i++)
    {
        det *= triang_mat[i][i];
    }

    return det;
}

__device__ float trace(const float matrix[][DIRECTIONS_MAX], const size_t rank)
{
    float sum_fdiag=0.f;

    // Iter through the first diagonal
    for(size_t i=0; i<rank; i++)
    {
        sum_fdiag += matrix[i][i];
    }

    return sum_fdiag;
}

__constant__ uchar2 noncorner_coords[MASK_MAX];

/* possible overhead: 
   * not enough registers 
   * uncoalesced memory access
   * unsufficient streaming multiprocessor warps
*/

__global__ void 
d_first_corner_measures(const float* im_templates, 
                        const size_t im_templates_pitch,
                        float* corner_measure,
                        const size_t corner_measure_pitch,
                        const size_t width, 
                        const size_t height, 
                        const size_t directions_n, 
                        const size_t filter_size, 
                        const float eps)
{    
    const size_t padding_size = filter_size/2; // floor division
    const size_t padding_size_twice = filter_size - 1;

    // NOTE: the use of ptrdiff_t is due to signed and unsigned conversions
    const size_t col_global = threadIdx.x + blockIdx.x * (blockDim.x - (ptrdiff_t)padding_size_twice);
    const size_t row_global = threadIdx.y + blockIdx.y * (blockDim.y - (ptrdiff_t)padding_size_twice);

    // Check if thread is outside the image "padded" region
    if (col_global >= width + padding_size_twice || row_global >= height + padding_size_twice)
        return;

    __shared__ float im_template_shr[DIRECTIONS_MAX][BLOCK_SIZE][BLOCK_SIZE];

    const ptrdiff_t col_global_shifted = col_global - (ptrdiff_t)padding_size;
    const ptrdiff_t row_global_shifted = row_global - (ptrdiff_t)padding_size;

    bool is_padding_zeros = col_global_shifted < 0 || col_global_shifted >= (ptrdiff_t)width || 
                            row_global_shifted < 0 || row_global_shifted >= (ptrdiff_t)height;

    if (is_padding_zeros) 
    {
        for (size_t direction_idx = 0; direction_idx < directions_n; direction_idx++)
        {
            im_template_shr[direction_idx][threadIdx.y][threadIdx.x] = 0.f;
        }
        return;
    }                    

    // Copy 'directions_n' tiles into shared memory
    for (size_t direction_idx = 0; direction_idx < directions_n; direction_idx++)
    {
        float val = *((float*)((char*)im_templates + (direction_idx * height + row_global_shifted) * im_templates_pitch) + col_global_shifted);
        im_template_shr[direction_idx][threadIdx.y][threadIdx.x] = val;
    }

    const ptrdiff_t col_local_shifted = threadIdx.x - (ptrdiff_t)padding_size;
    const ptrdiff_t row_local_shifted = threadIdx.y - (ptrdiff_t)padding_size;    

    bool is_padding = col_local_shifted < 0 || col_local_shifted >= blockDim.x - (ptrdiff_t)padding_size_twice || 
                      row_local_shifted < 0 || row_local_shifted >= blockDim.y - (ptrdiff_t)padding_size_twice;        

    if(is_padding)                      
        return;

    __syncthreads(); 

    float template_symmetric[DIRECTIONS_MAX][DIRECTIONS_MAX];
    // Initialise the result matrix
    for (size_t i = 0; i < directions_n; i++) 
    {
        for (size_t j = 0; j < directions_n; j++) 
        {
            template_symmetric[i][j] = 0.f;
        }
    }         

    const size_t mask_len = filter_size*filter_size - CORNER_NUM; 
    // Loop through the xy kernel
    // We have precomputed the valid kernel coord with corners removed in noncorner_coords
    for (size_t k = 0; k < mask_len; k++) {
        const size_t curr_row = row_local_shifted + noncorner_coords[k].y;
        const size_t curr_col = col_local_shifted + noncorner_coords[k].x;

        // Add the A * At contributions for this pixel
        // Avoid extra loopings by noting that the matrix is symmetrical, we will mirror it after
        for (size_t i = 0; i < directions_n; i++) 
        {
            for (size_t j = i; j < directions_n; j++) 
            {
                template_symmetric[i][j] += im_template_shr[i][curr_row][curr_col] * im_template_shr[j][curr_row][curr_col];
            }
        }
    }    

    // Mirror the matrix about the diagonal
    for (size_t i = 0; i < directions_n; i++) 
    {
        for (size_t j = 0; j < i; j++) 
        {
            template_symmetric[i][j] = template_symmetric[j][i];
        }
    }    

    const float trc = trace(template_symmetric, directions_n);
    // to save registers, the input matrix isn't coppied; thus, changed inplace
    const float det = determinant(template_symmetric, directions_n); 

    float *corner_measure_row = (float*)((char*)corner_measure + row_global_shifted * corner_measure_pitch);
    corner_measure_row[col_global_shifted] = det / (trc + eps);
}

// __device__ Npp32s nSrcStep;
// __device__ Npp32f *pSrc;

extern "C"
float* set_filter_src_image(const float *h_pSrc, 
                            const int width, 
                            const int height,
                            int &nSrcStep)
{
    // Npp32s nSrcStep_tmp;
    Npp32f *d_pSrc;

    d_pSrc = nppiMalloc_32f_C1(width, height, &nSrcStep); 
    checkCudaErrors(hipMemcpy2D(d_pSrc, 
                                 nSrcStep, 
                                 h_pSrc, 
                                 sizeof(float)*width, 
                                 sizeof(float)*width, 
                                 height, 
                                 hipMemcpyHostToDevice));   

    return d_pSrc;                                 
                                 
    // checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(pSrc), pSrc_tmp, width*height*nSrcStep_tmp, 0, hipMemcpyDeviceToDevice));
    // checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(&nSrcStep), &nSrcStep_tmp, sizeof(Npp32s), 0, hipMemcpyDeviceToDevice));
}

extern "C"
float* compute_templates(float* pSrc,
                         const int pSrcStep,
                         const int width, 
                         const int height, 
                         const float *conv_filter, 
                         const int filter_size)
{
    Npp32s nDstStep;

    Npp32f *pDst, *pKernel;
    float* im_template;                               

    hipMalloc((void**)&pKernel, sizeof(Npp32f)*filter_size*filter_size);
    hipMemcpy(pKernel, conv_filter, sizeof(Npp32f)*filter_size*filter_size, hipMemcpyHostToDevice);

    pDst = nppiMalloc_32f_C1(width, height, &nDstStep); 

    checkCudaErrors(hipHostMalloc(&im_template, sizeof(float) * width * height));

    NppiSize oSrcSize = {width, height};
    NppiPoint oSrcOffset = {0, 0};
    NppiSize oSizeROI = {width, height};
    NppiSize oKernelSize = {filter_size, filter_size};
    NppiPoint oAnchor = {filter_size/2, filter_size/2};

    // Npp32s nSrcStep_val;
    // Npp32f *pSrc_ptr;
    // checkCudaErrors(hipMemcpyFromSymbol(&nSrcStep_val, HIP_SYMBOL(nSrcStep), sizeof(Npp32s)));
    // checkCudaErrors(hipGetSymbolAddress((void**)&pSrc_ptr, pSrc));

    NPP_CHECK_NPP(nppiFilterBorder_32f_C1R(pSrc, 
                                           pSrcStep,
                                           oSrcSize,
                                           oSrcOffset,
                                           pDst,
                                           nDstStep,
                                           oSizeROI,
                                           pKernel,
                                           oKernelSize,
                                           oAnchor,
                                           NPP_BORDER_REPLICATE));

    // NPP_CHECK_NPP(nppiAbs_32f_C1R(pDst, nDstStep, pDst, nDstStep, oSizeROI));
                             
    checkCudaErrors(hipMemcpy2D(im_template, 
                                 sizeof(float)*width, 
                                 pDst, 
                                 nDstStep, 
                                 sizeof(Npp32f)*width, 
                                 height, 
                                 hipMemcpyDeviceToHost)); 

    // nppiFree(pSrc);
    nppiFree(pDst);
    nppiFree(pKernel);

    return im_template;                                    
}

extern "C"
void sequential_corner_measures()
{
}

extern "C"
int init_cuda_device(int argc, const char **argv)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        std::cerr << "CUDA error: no devices supporting CUDA." << std::endl;
        exit(EXIT_FAILURE);
    }

    int dev = findCudaDevice(argc, argv);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    std::cerr << "hipSetDevice GPU " << dev << " = " << deviceProp.name << std::endl;

    checkCudaErrors(hipSetDevice(dev));

    return dev;
}

void compute_noncorner_coords(size_t filter_size)
{
    size_t corner_indexes[] = {0, 1, filter_size-2, 
                               filter_size-1, filter_size, 2*filter_size-1, 
                               filter_size*(filter_size-2), filter_size*(filter_size-1)-1, filter_size*(filter_size-1),
                               filter_size*(filter_size-1)+1, filter_size*filter_size-2, filter_size*filter_size-1}; 

    uchar2 h_noncorner_coords[MASK_MAX];
    size_t noncorner_count = 0;
    for(unsigned i=0; i<filter_size; i++)
    {
        for(unsigned j=0; j<filter_size; j++)
        {
            bool is_corner = false;
            #pragma unroll
            for(auto corner_index : corner_indexes)
            {
                if(i * filter_size + j == corner_index)
                {
                    is_corner = true;
                    break;
                }
            }

            if(is_corner)
            {
                continue;
            }

            h_noncorner_coords[noncorner_count] = make_uchar2(i, j);
            noncorner_count += 1;
        }
    }

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(noncorner_coords), h_noncorner_coords, sizeof(uchar2)*noncorner_count));
}

/*
im_templates size: directions_n x width x height (flatten)
*/
extern "C"
float* first_corner_measures(const float *im_templates, 
                             const size_t width, 
                             const size_t height, 
                             const size_t directions_n, 
                             const size_t filter_size, 
                             const float eps)
{
    compute_noncorner_coords(filter_size);

    size_t d_im_templates_pitch, d_corner_measures_pitch;

    float *d_im_templates, *d_corner_measures, *h_corner_measures;

    checkCudaErrors(hipMallocPitch(&d_im_templates, 
                                    &d_im_templates_pitch, 
                                    sizeof(float) * width, 
                                    height * directions_n));
    checkCudaErrors(hipMemcpy2D(d_im_templates, 
                                 d_im_templates_pitch, 
                                 im_templates, 
                                 sizeof(float)*width, 
                                 sizeof(float)*width, 
                                 height * directions_n, 
                                 hipMemcpyHostToDevice));  
                                      
    checkCudaErrors(hipMallocPitch(&d_corner_measures, 
                                    &d_corner_measures_pitch, 
                                    sizeof(float) * width, 
                                    height));    
    checkCudaErrors(hipHostMalloc(&h_corner_measures, sizeof(float) * width * height));

    ptrdiff_t useful_region = BLOCK_SIZE - filter_size + 1;
    if(useful_region < 0)
    {
        printf("No useful region\n");
    }

    // int THREADS = 16;
    dim3 block_dim(BLOCK_SIZE,BLOCK_SIZE);
    dim3 grid_dim((width+useful_region-1)/useful_region, (height+useful_region-1)/useful_region);
    d_first_corner_measures<<<grid_dim, block_dim>>>(d_im_templates, 
                                                     d_im_templates_pitch, 
                                                     d_corner_measures, 
                                                     d_corner_measures_pitch, 
                                                     width, 
                                                     height, 
                                                     directions_n, 
                                                     filter_size, 
                                                     eps);
                                                     
    checkCudaErrors(hipMemcpy2D(h_corner_measures, 
                                  sizeof(float)*width, 
                                  d_corner_measures, 
                                  d_corner_measures_pitch, 
                                  sizeof(float)*width, 
                                  height, 
                                  hipMemcpyDeviceToHost)); 

    checkCudaErrors(hipFree(d_im_templates));
    checkCudaErrors(hipFree(d_corner_measures));

    return h_corner_measures;
}